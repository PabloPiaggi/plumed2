#include "hip/hip_runtime.h"
/* +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
   Copyright (c) 2011-2019 The plumed team
   (see the PEOPLE file at the root of the distribution for a list of names)

   See http://www.plumed.org for more information.

   This file is part of plumed, version 2.

   plumed is free software: you can redistribute it and/or modify
   it under the terms of the GNU Lesser General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   plumed is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with plumed.  If not, see <http://www.gnu.org/licenses/>.
+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
#include "Colvar.h"
#include "ActionRegister.h"

#include <string>
#include <cmath>
#include <stdio.h>

using namespace std;

namespace PLMD {
namespace gpucolvar {

//+PLUMEDOC COLVAR TEMPLATE2
/*
This file provides a template for if you want to introduce a new CV.

<!-----You should add a description of your CV here---->

\par Examples

<!---You should put an example of how to use your CV here--->

\plumedfile
# This should be a sample input.
t: TEMPLATE2 ATOMS=1,2
PRINT ARG=t STRIDE=100 FILE=COLVAR
\endplumedfile
<!---You should reference here the other actions used in this example--->
(see also \ref PRINT)

*/
//+ENDPLUMEDOC

class Template2 : public Colvar {
  bool pbc;

public:
  explicit Template2(const ActionOptions&);
// active methods:
  void calculate() override;
  static void registerKeywords(Keywords& keys);
};

__global__ void GPUFunction() {
  printf("Hello world from the GPU.\n");
}


PLUMED_REGISTER_ACTION(Template2,"TEMPLATE2")

void Template2::registerKeywords(Keywords& keys) {
  Colvar::registerKeywords(keys);
  keys.addFlag("TEMPLATE2_DEFAULT_OFF_FLAG",false,"flags that are by default not performed should be specified like this");
  keys.addFlag("TEMPLATE2_DEFAULT_ON_FLAG",true,"flags that are by default performed should be specified like this");
  //keys.add("compulsory","TEMPLATE2_COMPULSORY","all compulsory keywords should be added like this with a description here");
  keys.add("optional","TEMPLATE2_OPTIONAL","all optional keywords that have input should be added like a description here");
  keys.add("atoms","ATOMS","the keyword with which you specify what atoms to use should be added like this");
}

Template2::Template2(const ActionOptions&ao):
  PLUMED_COLVAR_INIT(ao),
  pbc(true)
{
  vector<AtomNumber> atoms;
  parseAtomList("ATOMS",atoms);
  if(atoms.size()!=2)
    error("Number of specified atoms should be 2");
  bool nopbc=!pbc;
  parseFlag("NOPBC",nopbc);
  pbc=!nopbc;
  checkRead();

  log.printf("  between atoms %d %d\n",atoms[0].serial(),atoms[1].serial());
  if(pbc) log.printf("  using periodic boundary conditions\n");
  else    log.printf("  without periodic boundary conditions\n");

  addValueWithDerivatives(); setNotPeriodic();

  requestAtoms(atoms);

  // function to run on the gpu
  GPUFunction<<<1, 1>>>();
  
  // kernel execution is asynchronous so sync on its completion
  hipDeviceSynchronize();
}

// calculator
void Template2::calculate() {

  Vector distance;
  if(pbc) {
    distance=pbcDistance(getPosition(0),getPosition(1));
  } else {
    distance=delta(getPosition(0),getPosition(1));
  }
  const double value=distance.modulo();
  const double invvalue=1.0/value;

  setAtomsDerivatives(0,-invvalue*distance);
  setAtomsDerivatives(1,invvalue*distance);
  setBoxDerivatives  (-invvalue*Tensor(distance,distance));
  setValue           (value);
}

}
}



